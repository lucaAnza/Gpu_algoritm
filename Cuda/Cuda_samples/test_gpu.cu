/* 

Programm to the test if GPU works

*/







#include <hip/hip_runtime.h>
#include<stdio.h>



// Funzione esegue la somma di n elementi
__global__ void test_gpu(int *arr , int n , int* sum_gpu ){
	
    int id = threadIdx.x;
	if(id<n){
		int add_factor = 12;
        arr[id] = add_factor;
		atomicAdd( sum_gpu , add_factor);
	}

}
		
int main(){
	

	int n = 5; // number of threads
	int *GpuArr;
	int CpuArr[n];
	int* sum_gpu;
	int sum_cpu = 0;
	hipMalloc(&GpuArr , sizeof(int) * n);
	hipMalloc(&sum_gpu , sizeof(int) * 1);

	test_gpu<<<1,n>>>(GpuArr , n , sum_gpu);
	
	hipMemcpy(CpuArr , GpuArr , sizeof(int) * n , hipMemcpyDeviceToHost );
	hipMemcpy(&sum_cpu , sum_gpu , sizeof(int) * 1 , hipMemcpyDeviceToHost );
	hipDeviceSynchronize();
	
	int sumTest = 0;
	for ( int i=0 ; i<n ; i++){
		sumTest += CpuArr[i];
	}

	if(sumTest == sum_cpu ){
		printf("Hey la Gpu funziona a dovere!\nBuon lavoro!\n");
	}else{
		printf("ERROR : Comportamento anomalo controllare lo stato della Gpu!");
	}

	return 0;

}